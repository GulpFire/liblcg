/******************************************************
 * C++ Library of the Linear Conjugate Gradient Methods (LibLCG)
 * 
 * Copyright (C) 2022  Yi Zhang (yizhang-geo@zju.edu.cn)
 * 
 * LibLCG is distributed under a dual licensing scheme. You can
 * redistribute it and/or modify it under the terms of the GNU Lesser
 * General Public License (LGPL) as published by the Free Software Foundation,
 * either version 2 of the License, or (at your option) any later version. 
 * You should have received a copy of the GNU Lesser General Public 
 * License along with this program. If not, see <http://www.gnu.org/licenses/>. 
 * 
 * If the terms and conditions of the LGPL v.2. would prevent you from
 * using the LibLCG, please consider the option to obtain a commercial
 * license for a fee. These licenses are offered by the LibLCG developing 
 * team. As a rule, licenses are provided "as-is", unlimited in time for 
 * a one time fee. Please send corresponding requests to: yizhang-geo@zju.edu.cn. 
 * Please do not forget to include some description of your company and the 
 * realm of its activities. Also add information on how to contact you by 
 * electronic and paper mail.
 ******************************************************/

#include "cmath"
#include "ctime"
#include "iostream"

#include "clcg_cuda.h"


typedef int (*cuda_solver_ptr)(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg_symmetric(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_cuda(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, const hipDoubleComplex* B, 
    const int n_size, const int nz_size, const clcg_para* param, void* instance, hipblasHandle_t cub_handle, 
    hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_solver_ptr cg_solver;
    switch (solver_id)
	{
		case CLCG_BICG:
			cg_solver = clbicg;
			break;
		case CLCG_BICG_SYM:
			cg_solver = clbicg_symmetric;
			break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cg_solver(Afp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

typedef int (*cuda_precondtioned_solver_ptr)(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, 
    hipDoubleComplex* m, const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, 
    void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clpcg(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_preconditioned_cuda(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, 
    hipDoubleComplex* m, const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_precondtioned_solver_ptr cgp_solver;
    switch (solver_id)
	{
		case CLCG_PCG:
			cgp_solver = clpcg; break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cgp_solver(Afp, Mfp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

int clbicg(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipDoubleComplex *d_m = nullptr, *d_B = nullptr;
	hipDoubleComplex *r1k = nullptr, *r2k = nullptr;
	hipDoubleComplex *d1k = nullptr, *d2k = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d_B, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&r1k, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&r2k, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&d1k, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d2k, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_d1k, dvec_d2k, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_d1k, n_size, d1k, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_d2k, n_size, d2k, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipDoubleComplex ak, nak, conj_ak, Ad1d2, r1r2_next, betak, conj_betak;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(r1k, d_B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, r1k, 1); // r0 -= Ax
    hipMemcpy(d1k, r1k, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // d0 = r0

	clcg_vecZ_conjugate(r1k, r2k, n_size);
	hipMemcpy(d2k, r2k, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

	hipDoubleComplex r1r2;
    hipblasZdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2);

	lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, r1k, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_d1k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasZdotc(cub_handle, n_size, d2k, 1, Ax, 1, &Ad1d2);
        ak = hipCdiv(r1r2, Ad1d2);
        nak = hipCmul(none, ak);
		conj_ak = hipConj(nak);

        hipblasZaxpy(cub_handle, n_size, &ak, d1k, 1, d_m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, r1k, 1);

        if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, r1k, 1, &rk_mod);

		Afp(instance, cub_handle, cus_handle, dvec_d2k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE);

		hipblasZaxpy(cub_handle, n_size, &conj_ak, Ax, 1, r2k, 1);

		hipblasZdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2_next);
		betak = hipCdiv(r1r2_next, r1r2);
		conj_betak = hipConj(betak);
		r1r2 = r1r2_next;

        hipblasZscal(cub_handle, n_size, &betak, d1k, 1);
        hipblasZaxpy(cub_handle, n_size, &one, r1k, 1, d1k, 1);

		hipblasZscal(cub_handle, n_size, &conj_betak, d2k, 1);
        hipblasZaxpy(cub_handle, n_size, &one, r2k, 1, d2k, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(r1k);
		hipFree(r2k);
		hipFree(d1k);
		hipFree(d2k);	
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_d1k);
		hipsparseDestroyDnVec(dvec_d2k);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clbicg_symmetric(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipDoubleComplex *d_m = nullptr, *d_B = nullptr;
	hipDoubleComplex *rk = nullptr, *dk = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d_B, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&rk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&dk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_dk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipDoubleComplex ak, nak, rkrk2, betak, dkAx;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, d_B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax
    hipMemcpy(dk, rk, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // d0 = r0

	hipDoubleComplex rkrk;
    hipblasZdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk);

	lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasZdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
        ak = hipCdiv(rkrk, dkAx);
        nak = hipCmul(none, ak);

        hipblasZaxpy(cub_handle, n_size, &ak, dk, 1, d_m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

        if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

		hipblasZdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk2);
		betak = hipCdiv(rkrk2, rkrk);
		rkrk = rkrk2;

        hipblasZscal(cub_handle, n_size, &betak, dk, 1);
        hipblasZaxpy(cub_handle, n_size, &one, rk, 1, dk, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(rk);
		hipFree(dk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clpcg(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipDoubleComplex *d_m = nullptr, *d_B = nullptr;
    hipDoubleComplex *rk = nullptr, *dk = nullptr, *sk = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d_B, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&rk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&dk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&sk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_rk, dvec_dk, dvec_sk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_64F);
    hipsparseCreateDnVec(&dvec_rk, n_size, rk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_64F);
    hipsparseCreateDnVec(&dvec_sk, n_size, sk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
    hipDoubleComplex ak, nak, d_old, betak, dkAx;

    Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, d_B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax

	Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_dk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	hipDoubleComplex d_new;
    hipblasZdotu(cub_handle, n_size, rk, 1, dk, 1, &d_new);

    lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

    int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        hipblasZdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
		ak = hipCdiv(d_new, dkAx);
        nak = hipCmul(none, ak);

        hipblasZaxpy(cub_handle, n_size, &ak, dk, 1, d_m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

		if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

        Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_sk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

		d_old = d_new;
        hipblasZdotu(cub_handle, n_size, rk, 1, sk, 1, &d_new);

		betak = hipCdiv(d_new, d_old);

        hipblasZscal(cub_handle, n_size, &betak, dk, 1);
        hipblasZaxpy(cub_handle, n_size, &one, sk, 1, dk, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(rk);
		hipFree(dk);
		hipFree(sk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_rk);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_sk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}