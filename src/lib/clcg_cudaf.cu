/******************************************************
 * C++ Library of the Linear Conjugate Gradient Methods (LibLCG)
 * 
 * Copyright (C) 2022  Yi Zhang (yizhang-geo@zju.edu.cn)
 * 
 * LibLCG is distributed under a dual licensing scheme. You can
 * redistribute it and/or modify it under the terms of the GNU Lesser
 * General Public License (LGPL) as published by the Free Software Foundation,
 * either version 2 of the License, or (at your option) any later version. 
 * You should have received a copy of the GNU Lesser General Public 
 * License along with this program. If not, see <http://www.gnu.org/licenses/>. 
 * 
 * If the terms and conditions of the LGPL v.2. would prevent you from
 * using the LibLCG, please consider the option to obtain a commercial
 * license for a fee. These licenses are offered by the LibLCG developing 
 * team. As a rule, licenses are provided "as-is", unlimited in time for 
 * a one time fee. Please send corresponding requests to: yizhang-geo@zju.edu.cn. 
 * Please do not forget to include some description of your company and the 
 * realm of its activities. Also add information on how to contact you by 
 * electronic and paper mail.
 ******************************************************/

#include "cmath"
#include "ctime"
#include "iostream"

#include "clcg_cudaf.h"


typedef int (*cuda_solver_ptr)(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg_symmetric(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_cuda(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, const hipComplex* B, 
    const int n_size, const int nz_size, const clcg_para* param, void* instance, hipblasHandle_t cub_handle, 
    hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_solver_ptr cg_solver;
    switch (solver_id)
	{
		case CLCG_BICG:
			cg_solver = clbicg;
			break;
		case CLCG_BICG_SYM:
			cg_solver = clbicg_symmetric;
			break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cg_solver(Afp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

typedef int (*cuda_precondtioned_solver_ptr)(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, 
    hipComplex* m, const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, 
    void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clpcg(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_preconditioned_cuda(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, 
    hipComplex* m, const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_precondtioned_solver_ptr cgp_solver;
    switch (solver_id)
	{
		case CLCG_PCG:
			cgp_solver = clpcg; break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cgp_solver(Afp, Mfp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

int clbicg(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipComplex *d_m = nullptr, *d_B = nullptr;
	hipComplex *r1k = nullptr, *r2k = nullptr;
	hipComplex *d1k = nullptr, *d2k = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipComplex));
	hipMalloc(&d_B, n_size * sizeof(hipComplex));
    hipMalloc(&r1k, n_size * sizeof(hipComplex));
	hipMalloc(&r2k, n_size * sizeof(hipComplex));
    hipMalloc(&d1k, n_size * sizeof(hipComplex));
	hipMalloc(&d2k, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_d1k, dvec_d2k, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_d1k, n_size, d1k, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_d2k, n_size, d2k, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipComplex ak, nak, conj_ak, Ad1d2, r1r2_next, betak, conj_betak;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(r1k, d_B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, r1k, 1); // r0 -= Ax
    hipMemcpy(d1k, r1k, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // d0 = r0

	clcg_vecC_conjugate(r1k, r2k, n_size);
	hipMemcpy(d2k, r2k, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice);

	hipComplex r1r2;
    hipblasCdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2);

	float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, r1k, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_d1k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasCdotc(cub_handle, n_size, d2k, 1, Ax, 1, &Ad1d2);
        ak = hipCdivf(r1r2, Ad1d2);
        nak = hipCmulf(none, ak);
		conj_ak = hipConjf(nak);

        hipblasCaxpy(cub_handle, n_size, &ak, d1k, 1, d_m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, r1k, 1);

        if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, r1k, 1, &rk_mod);

		Afp(instance, cub_handle, cus_handle, dvec_d2k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE);

		hipblasCaxpy(cub_handle, n_size, &conj_ak, Ax, 1, r2k, 1);

		hipblasCdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2_next);
		betak = hipCdivf(r1r2_next, r1r2);
		conj_betak = hipConjf(betak);
		r1r2 = r1r2_next;

        hipblasCscal(cub_handle, n_size, &betak, d1k, 1);
        hipblasCaxpy(cub_handle, n_size, &one, r1k, 1, d1k, 1);

		hipblasCscal(cub_handle, n_size, &conj_betak, d2k, 1);
        hipblasCaxpy(cub_handle, n_size, &one, r2k, 1, d2k, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(r1k);
		hipFree(r2k);
		hipFree(d1k);
		hipFree(d2k);	
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_d1k);
		hipsparseDestroyDnVec(dvec_d2k);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clbicg_symmetric(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipComplex *d_m = nullptr, *d_B = nullptr;
	hipComplex *rk = nullptr, *dk = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d_B, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&rk, n_size * sizeof(hipComplex));
    hipMalloc(&dk, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_dk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipComplex ak, nak, rkrk2, betak, dkAx;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, d_B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax
    hipMemcpy(dk, rk, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // d0 = r0

	hipComplex rkrk;
    hipblasCdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk);

	float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasCdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
        ak = hipCdivf(rkrk, dkAx);
        nak = hipCmulf(none, ak);

        hipblasCaxpy(cub_handle, n_size, &ak, dk, 1, d_m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

        if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

		hipblasCdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk2);
		betak = hipCdivf(rkrk2, rkrk);
		rkrk = rkrk2;

        hipblasCscal(cub_handle, n_size, &betak, dk, 1);
        hipblasCaxpy(cub_handle, n_size, &one, rk, 1, dk, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(rk);
		hipFree(dk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clpcg(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipComplex *d_m = nullptr, *d_B = nullptr;
    hipComplex *rk = nullptr, *dk = nullptr, *sk = nullptr, *Ax = nullptr;
	hipMalloc(&d_m, n_size * sizeof(hipComplex));
    hipMalloc(&d_B, n_size * sizeof(hipComplex));
    hipMalloc(&rk, n_size * sizeof(hipComplex));
    hipMalloc(&dk, n_size * sizeof(hipComplex));
    hipMalloc(&sk, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

	// Copy initial solutions
	hipMemcpy(d_m, m, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n_size * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipsparseDnVecDescr_t dvec_m, dvec_rk, dvec_dk, dvec_sk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, d_m, HIP_C_32F);
    hipsparseCreateDnVec(&dvec_rk, n_size, rk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_32F);
    hipsparseCreateDnVec(&dvec_sk, n_size, sk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
    hipComplex ak, nak, d_old, betak, dkAx;

    Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, d_B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax

	Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_dk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	hipComplex d_new;
    hipblasCdotu(cub_handle, n_size, rk, 1, dk, 1, &d_new);

    float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

    int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, d_m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, d_m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        hipblasCdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
		ak = hipCdivf(d_new, dkAx);
        nak = hipCmulf(none, ak);

        hipblasCaxpy(cub_handle, n_size, &ak, dk, 1, d_m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

		if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, d_m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

        Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_sk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

		d_old = d_new;
        hipblasCdotu(cub_handle, n_size, rk, 1, sk, 1, &d_new);

		betak = hipCdivf(d_new, d_old);

        hipblasCscal(cub_handle, n_size, &betak, dk, 1);
        hipblasCaxpy(cub_handle, n_size, &one, sk, 1, dk, 1);
	}

	func_ends:
	{
		// Copy to host memories
		hipMemcpy(m, d_m, n_size * sizeof(hipComplex), hipMemcpyDeviceToHost);

		hipFree(d_m);
		hipFree(d_B);
		hipFree(rk);
		hipFree(dk);
		hipFree(sk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_rk);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_sk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}